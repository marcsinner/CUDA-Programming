#include "driver.h"
#include "kernels/kernels.h"
#include "util.hpp"
#include <chrono>
#include <hipblas.h>

float compute(std::vector<float> &C, const std::vector<float> &A, const std::vector<float> &B,
              const Configuration &config) {

    hipDeviceReset();
    CHECK_ERR;

    float *devA{nullptr}, *devB{nullptr}, *devC{nullptr};
    {
	hipMalloc(&devA, A.size()* sizeof(float)); CHECK_ERR;
	hipMalloc(&devB, B.size()* sizeof(float)); CHECK_ERR;
	hipMalloc(&devC, C.size()* sizeof(float)); CHECK_ERR;
	

 // TODO: Allocate matrices A, B, C on device
    }

    {

	hipMemcpy(devA,A.data(),A.size()* sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;
	hipMemcpy(devB,B.data(),B.size()* sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;
	hipMemcpy(devC,C.data(),C.size()* sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;



      // TODO: Copy the data from host to the device
      // NOTE: You may copy C as well, as it is zeroed, or hipMemset it to zero on the device
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float cpuTime{};
    hipEvent_t startTimer{}, stopTimer{};
    hipEventCreate(&startTimer);
    hipEventCreate(&stopTimer);


    // Start computing
    hipEventRecord(startTimer, 0);
    switch (config.kernelType) {
        case KernelType::KERNEL_CPU: {
            std::chrono::high_resolution_clock::time_point begin =
                std::chrono::high_resolution_clock::now();
            for (int i = 0; i < config.numRepeats; ++i) {
                cpu::matrixMult(C, A, B, config);
            }
            std::chrono::high_resolution_clock::time_point end =
                std::chrono::high_resolution_clock::now();
            cpuTime =
                std::chrono::duration_cast<std::chrono::duration<double>>(end - begin).count();
            break;
        }
        case KernelType::KERNEL_CUBLAS: {
            float alpha = 1.0f, beta = 1.0f;
            for (int i = 0; i < config.numRepeats; ++i) {
                hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, config.matrixSize, config.matrixSize,
                            config.matrixSize, &alpha, devA, config.matrixSize, devB,
                            config.matrixSize, &beta, devC, config.matrixSize);
            }
            CHECK_ERR;
            break;
        }
        default: {
            // NOTE: repeat loop is inside of gpu::matrixMult
            gpu::matrixMult(devA, devB, devC, config);
            break;
        }
    }

    hipEventRecord(stopTimer, 0);
    hipEventSynchronize(stopTimer);
    CHECK_ERR;
    // NOTE: hipEventSynchronize(stopTimer) is implicit hipDeviceSynchronize() in this context

    float gpuTime{};
    hipEventElapsedTime(&gpuTime, startTimer, stopTimer);

    // release resources
    hipblasDestroy(handle);
    hipEventDestroy(startTimer);
    hipEventDestroy(stopTimer);

    {
 
	hipMemcpy(const_cast<float *>(C.data()), devC,C.size() * sizeof(float), hipMemcpyDeviceToHost);CHECK_ERR;
 
// TODO: transfer matrix C back, from device to the host
    }

    {
 	hipFree(devA);CHECK_ERR;
	hipFree(devB);CHECK_ERR;
	hipFree(devC);CHECK_ERR;
	
   
  // TODO: clean gpu memory
    }
    return (config.kernelType == KernelType::KERNEL_CPU) ? cpuTime : gpuTime;
}
